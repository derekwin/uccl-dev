#include "hip/hip_runtime.h"


__global__ void persistent_worker(TaskQueue* q, int worker_id) {
  while (true) {
    int task_id = q->acquire_task(worker_id);
    if (task_id < 0) {
      if (q->should_exit()) break;
      continue;
    }

    // 执行任务
    Task t = q->tasks[task_id];
    process_task(t);

    // 标记完成
    q->complete_task(worker_id, task_id);
  }
}
